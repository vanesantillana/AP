
#include <hip/hip_runtime.h>
#include <stdio.h>      
#include <stdlib.h>     
#include <time.h>
#include <iostream>

using namespace std;

__global__
void sumaMatrixKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n*n)
		C[i] = A[i] +B[i];
}

__global__
void sumaMatrixKernelRow(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		for(int j=0;j<n;j++)
			 C[i*n+j] = A[i*n+j] + B[i*n+j];
	}	
}

__global__
void sumaMatrixKernelColumn(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		for(int j=0;j<n;j++)
			  C[i+j*n] = A[i+j*n] + B[i+j*n];

	}
	
}

void sumaMatrix(float* A, float* B, float* C, int tam)
{
	int size = (tam*tam) * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,size);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	sumaMatrixKernel<<<ceil(tam*tam/256.0),256>>>(d_A,d_B,d_C,tam);
	//sumaMatrixKernelRow<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);
	//sumaMatrixKernelColumn<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);

	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	
}

void datosRandom(float *array,int n){
	//srand(time(NULL));
	for(int i = 0; i < n; i++){
	    for(int j = 0; j < n; j++)
	    	array[i*n+j] = 1;
	}
}

void printMatrix(float *array,int n){
  for(int i = 0; i < n; i++){
    printf("%f ", array[i]);
  }
  printf("\n");
}

int main()
{
	int n = 10;
	float *h_A,*h_B,*h_C;
	h_A = new float[n*n];
	h_B = new float[n*n];
	h_C = new float[n*n];

	datosRandom(h_A,n);
	datosRandom(h_B,n);
    sumaMatrix(h_A,h_B,h_C,n);
    printMatrix(h_C,n);

   	return 0;	
}
#include "hip/hip_runtime.h"
#include <stdio.h>     
#include <stdlib.h>    
#include <time.h>
#include <iostream>

using namespace std;

__global__
void matrixVectorKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		C[i] = 0;
		for(int j=0;j<n;j++)
			 C[i] += A[i*n+j] * B[j];
	}
	
}



void matrixVector(float* A, float* B, float* C, int tam)
{
	int sizeA = (tam*tam) * sizeof(float);
	int size =  tam * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,sizeA);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	matrixVectorKernel<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	
}

void datosRandom(float *array){
	for(int i = 0; i < n; i++)
	{
	   for(int j = 0; j < n; j++)
	     array[i*n+j] = rand() % 100;
    }
}
void printMatrix(float *array){
	for(int i = 0; i < n; i++){
    	cout<<h_A[i]<<" ";
    }
    cout<<endl;
}

int main()
{
	int n = 3;
	float *h_A,*h_B,*h_C;
	h_A = new float[n*n];
	h_B = new float[n*n];
	h_C = new float[n*n];

	datosRandom(h_A);
	datosRandom(h_B);
    matrixVector(h_A,h_B,h_C,n);
    printMatrix(h_C);

   	return 0;
}
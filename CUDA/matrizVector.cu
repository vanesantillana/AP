
#include <hip/hip_runtime.h>
#include <stdio.h>     
#include <stdlib.h>    
#include <time.h>
#include <iostream>

using namespace std;

__global__
void matrixVectorKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		C[i] = 0;
		for(int j=0;j<n;j++)
			 C[i] += A[i*n+j] * B[j];
	}
	
}



void matrixVector(float* A, float* B, float* C, int tam)
{
	int sizeA = (tam*tam) * sizeof(float);
	int size =  tam * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,sizeA);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	matrixVectorKernel<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	
}



void datosRandom(float *array,int n){
	//srand(time(NULL));
	for(int i = 0; i < n; i++){
	    for(int j = 0; j < n; j++)
	    	array[i*n+j] = 1;
	}
}

void printMatrix(float *array,int n){
  for(int i = 0; i < n; i++){
    printf("%f ", array[i]);
  }
  printf("\n");
}
int main()
{
	int n = 10;
	float *h_A,*h_B,*h_C;
	h_A = new float[n*n];
	h_B = new float[n*n];
	h_C = new float[n*n];

	datosRandom(h_A,n);
	datosRandom(h_B,n);
    matrixVector(h_A,h_B,h_C,n);
    printMatrix(h_C,n);

   	return 0;
}
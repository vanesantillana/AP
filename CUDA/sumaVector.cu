#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>    
#include <time.h>
#include <iostream>

using namespace std;

__global__
void vecAddKernel(float *A, float *B, float *C, int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n) 
    C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n){
  int size = n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  vecAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void datosRandom(float *array){
  for(int i = 0; i < n; i++)
  {
     for(int j = 0; j < n; j++)
       array[i*n+j] = rand() % 100;
    }
}

void printMatrix(float *array){
  for(int i = 0; i < n; i++){
      cout<<h_A[i]<<" ";
    }
    cout<<endl;
}

int main(){
  int n,i;
  float *h_A,*h_B,*h_C;
  scanf("%d", &n);
  h_A = (float*) malloc(n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  
  datosRandom(h_A);
  datosRandom(h_B);
  vecAdd(h_A,h_B,h_C,n);
  printMatrix(h_C);

  return 0;
}

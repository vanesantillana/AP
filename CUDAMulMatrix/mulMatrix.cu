
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 16

__global__
void MatrixMultKernel(float *d_M, float *d_N, float *d_P, int Width){
  int Row = blockIdx.y*blockDim.y + threadIdx.y;
  int Col = blockIdx.x*blockDim.x + threadIdx.x;

  if(Row < Width && Col < Width){
      float Pvalue = 0;
      for(int k = 0; k < Width; ++k){
          Pvalue += d_M[Row*Width + k] * d_N[k*Width+Col];
      }
      d_P[Row*Width+Col] = Pvalue;
  }
}

void MatrixMult(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  int NumBlocks = n/BLOCK_WIDTH;
  if (n % BLOCK_WIDTH) NumBlocks++;
  
  dim3 dimGrid(NumBlocks,NumBlocks);
  dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH);

  MatrixMultKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void datosRandom(float *array,int n){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < n; j++)
      array[i*n+j] = rand() % 100;
  }
}

void printMatrix(float *array,int n){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < n; j++){
        printf("%f ", array[i*n+j]);
    }
    printf("\n"); 
  }  
}

int main(){
  int n=10;
  float *h_A,*h_B,*h_C;

  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*n*sizeof(float));
  h_C = (float*) malloc(n*n*sizeof(float));
  
  datosRandom(h_A,n);
  datosRandom(h_B,n);

  MatrixMult(h_A,h_B,h_C,n);
  printMatrix(h_C,n);
  
  return 0;
}
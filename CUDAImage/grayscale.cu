#include "hip/hip_runtime.h"
#include <stdio.h>     
#include <stdlib.h>    

#include "opencv/cv.h"
#include "opencv/highgui.h"
#include "opencv/imgproc/imgproc.hpp"

#define CHANNELS 3
__global__
void colorConvert(unsigned char* grayImage, unsigned char* rgbImage, int width , int height)
{
	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	if(x< width && y<height)
	{
		int grayOffset= y*width+x;
		int rgbOffset = grayOffset*CHANNELS;
		unsigned char r = rgbImage[rgbOffset ];
		unsigned char g = rgbImage[rgbOffset+2];
		unsigned char b = rgbImage[rgbOffset+3];
		grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
	
}


void rgbToGray(unsigned char* grayImage, unsigned char* rgbImage, int width , int height)
{
	int sizeimage = (width*height) * sizeof(unsigned char);

	unsigned char *d_rgbImage,*d_grayImage;

	hipMalloc((void**)&d_rgbImage,sizeimage);
	//hipMalloc((void**)&d_grayImage,sizeimage);

	hipMemcpy(d_rgbImage,rgbImage,sizeimage,hipMemcpyHostToDevice);

	colorConvert<<<ceil(width*height/256.0),256>>>(d_grayImage,d_rgbImage,width,height);

	hipMemcpy(grayImage,d_grayImage,sizeimage,hipMemcpyDeviceToHost);

	hipFree(d_grayImage);hipFree(d_rgbImage);
	
}


int main()
{
	char* imageName = argv[1];

	Mat image;
	image = imread( imageName, 1 );

	if( argc != 2 || !image.data )
	{
	  printf( " No image data \n " );
	  return -1;
	}

	Mat gray_image;
	int width = image->width;
    int height = image->height;
	//cvtColor( image, gray_image, CV_BGR2GRAY );
	rgbToGray(gray_image,image,width,height);

	imwrite( "paisajegray.png", gray_image );

 	namedWindow( imageName, CV_WINDOW_AUTOSIZE );
 	namedWindow( "Gray image", CV_WINDOW_AUTOSIZE );

	imshow( imageName, image );
 	imshow( "Gray image", gray_image );

 	waitKey(0);

   	return 0;
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <stdlib.h>
#include <iostream>
#include "lodepng.h"
using namespace std;

__global__
void PictureKernell(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int new_pos;
  
  if((y < n) && (x < m)) {
    new_pos = (y*m+x)*4;
    unsigned char r = d_Pin[new_pos];
    unsigned char g = d_Pin[new_pos+1];
    unsigned char b = d_Pin[new_pos+2];

    d_Pout[new_pos] = 0.21f*r + 0.71f*g + 0.07f*b;
    d_Pout[new_pos+1] = d_Pout[new_pos];
    d_Pout[new_pos+2] = d_Pout[new_pos];
    d_Pout[new_pos+3] = d_Pin[new_pos+3];
  }
}

__global__
void PictureKernel1D(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  x = x*4;
  if(x < n*m*4) {
    unsigned char r = d_Pin[x];
    unsigned char g = d_Pin[x+1];
    unsigned char b = d_Pin[x+2];
    d_Pout[x] = 0.21f*r + 0.71f*g + 0.07f*b;
    d_Pout[x+1] = d_Pout[x];
    d_Pout[x+2] = d_Pout[x];
    d_Pout[x+3] = d_Pin[x+3];
  }
}

void Picture(unsigned char* Pin, unsigned char* Pout, int n, int m){
  unsigned char* d_Pout, *d_Pin;
  long int size = n*m*4;
  hipMalloc((void **) &d_Pin,size);
  hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_Pout,size);

  dim3 gridDim((m-1)/8+1,(n-1)/16+1,1);
  dim3 blockDim(8,16,1);
  PictureKernell<<<gridDim,blockDim>>>(d_Pin,d_Pout,n,m);
  //PictureKernel1D<<<(size-1)/256+1,256>>>(d_Pin,d_Pout,n,m);
  hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
  hipFree(d_Pin); hipFree(d_Pout);
}

int main(int argc, char * argv[] ){
  unsigned char *image, *out_image;
  int i;
  char name_in[100], name_out[100];
  unsigned width, height;
  if(argv[1] == NULL or argv[2] == NULL)
    cout << "Usage\n inverse.cu [input image] [output image]\n";
  strcpy(name_in,argv[1]);
  strcpy(name_out,argv[2]);
  i = lodepng_decode32_file(&image, &width, &height, name_in);
  if(i < 0) printf("NO\n");
  out_image = (unsigned char*) malloc(width*height*4);
  Picture(image,out_image,height,width);
  lodepng_encode32_file(name_out,out_image,width,height);

  free(image);
  free(out_image);
  return 0;
}
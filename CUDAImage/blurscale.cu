#include "hip/hip_runtime.h"
#include <stdio.h>     
#include <stdlib.h>    

#include "opencv/cv.h"
#include "opencv/highgui.h"
#include "opencv/imgproc/imgproc.hpp"

#define CHANNELS 3
void blurKernel(unsigned char * in, unsigned char * out, int w, int h) {
      int Col  = blockIdx.x * blockDim.x + threadIdx.x;
      int Row  = blockIdx.y * blockDim.y + threadIdx.y;

      if (Col < w && Row < h) {
          int pixVal = 0;
          int pixels = 0;

          for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
              for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {

                  int curRow = Row + blurRow;
                  int curCol = Col + blurCol;

                  if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                      pixVal += in[curRow * w + curCol];
                      pixels++; 
                  }
              }
          }
          out[Row * w + Col] = (unsigned char)(pixVal / pixels);
      }
  }


void rgbToGray(unsigned char* grayImage, unsigned char* rgbImage, int width , int height)
{
	int sizeimage = (width*height) * sizeof(unsigned char);

	unsigned char *d_rgbImage,*d_grayImage;

	hipMalloc((void**)&d_rgbImage,sizeimage);
	//hipMalloc((void**)&d_grayImage,sizeimage);

	hipMemcpy(d_rgbImage,rgbImage,sizeimage,hipMemcpyHostToDevice);

	colorConvert<<<ceil(width*height/256.0),256>>>(d_grayImage,d_rgbImage,width,height);

	hipMemcpy(grayImage,d_grayImage,sizeimage,hipMemcpyDeviceToHost);

	hipFree(d_grayImage);hipFree(d_rgbImage);
	
}


int main()
{
	char* imageName = argv[1];

	Mat image;
	image = imread( imageName, 1 );

	if( argc != 2 || !image.data )
	{
	  printf( " No image data \n " );
	  return -1;
	}

	Mat gray_image;
	int width = image->width;
    int height = image->height;
	//cvtColor( image, gray_image, CV_BGR2GRAY );
	rgbToGray(gray_image,image,width,height);

	imwrite( "paisajegray.png", gray_image );

 	namedWindow( imageName, CV_WINDOW_AUTOSIZE );
 	namedWindow( "Gray image", CV_WINDOW_AUTOSIZE );

	imshow( imageName, image );
 	imshow( "Gray image", gray_image );

 	waitKey(0);

   	return 0;
}